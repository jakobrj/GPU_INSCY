#include "hip/hip_runtime.h"
#include "util.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <numeric>

#include <ATen/ATen.h>
#include <torch/extension.h>
#include "nvToolsExt.h"
#include "TmpMalloc.cuh"

#define SECTION_SIZE 64
#define SECTION_SIZE_LARGE 512
#define BLOCK_WIDTH 64

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


using namespace std;

float *copy_to_device(vector <vector<float>> X, int number_of_points, int number_of_dims) {
    float *d_X;
    hipMalloc(&d_X, sizeof(float) * number_of_points * number_of_dims);
    for (int i = 0; i < number_of_points; i++) {
        float *h_x_i = X[i].data();
        hipMemcpy(&d_X[i * number_of_dims], h_x_i, sizeof(float) * number_of_dims, hipMemcpyHostToDevice);
    }
    return d_X;
}


float *copy_to_device(at::Tensor X, int number_of_points, int number_of_dims) {
    float *d_X;
    hipMalloc(&d_X, sizeof(float) * number_of_points * number_of_dims);
    hipMemcpy(d_X, X.data_ptr<float>(), sizeof(float) * number_of_points * number_of_dims, hipMemcpyHostToDevice);
    return d_X;
}

__global__
void print_array_gpu(int *x, int n) {
    for (int i = 0; i < n; i++) {
        if (x[i] < 10 && x[i] > -1)
            printf(" ");
        if (x[i] < 100 && x[i] > -10)
            printf(" ");
        printf("%d ", x[i]);
    }
    printf("\n");
}

__global__
void print_array_gpu(float *x, int n) {
    for (int i = 0; i < n; i++) {
        printf("%f ", x[i]);
    }
    printf("\n");
}

__global__
void print_array_gpu(bool *x, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d ", (int) x[i]);
    }
    printf("\n");
}

__global__
void scan_kernel_eff(int *x, int *y, int n) {
/**
 * from the cuda book
 */
    __shared__ int XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        XY[threadIdx.x] = x[i];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = SECTION_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < n) {
        y[i] = XY[threadIdx.x];
    }
}


__global__
void scan_kernel_eff_large1(int *x, int *y, int *S, int n) {
/**
 * from the cuda book
 */
    __shared__ int XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        XY[threadIdx.x] = x[i];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = SECTION_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < n) {
        y[i] = XY[threadIdx.x];
    }

    if (threadIdx.x == 0) {
        S[blockIdx.x] = XY[SECTION_SIZE - 1];
    }

}

__global__
void scan_kernel_eff_large3(int *y, int *S, int n) {
/**
 * from the cuda book
 */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x > 0 && i < n) {
        y[i] += S[blockIdx.x - 1];
    }
}

void inclusive_scan(int *source, int *result, int n) {
    int numBlocks = n / SECTION_SIZE;
    if (n % SECTION_SIZE) numBlocks++;

    if (n > SECTION_SIZE) {
        int *S;
        hipMalloc((void **) &S, numBlocks * sizeof(int));
        scan_kernel_eff_large1 << < numBlocks, SECTION_SIZE >> > (source, result, S, n);
        inclusive_scan(S, S, numBlocks);
        scan_kernel_eff_large3 << < numBlocks, SECTION_SIZE >> > (result, S, n);
        hipFree(S);
    } else {
        scan_kernel_eff << < numBlocks, SECTION_SIZE >> > (source, result, n);
    }
}

void inclusive_scan_points(int *source, int *result, int n, TmpMalloc *tmps) {
    int numBlocks = n / SECTION_SIZE;
    if (n % SECTION_SIZE) numBlocks++;

    if (n > SECTION_SIZE) {
        int *S = tmps->malloc_points();
        scan_kernel_eff_large1 << < numBlocks, SECTION_SIZE >> > (source, result, S, n);
        inclusive_scan(S, S, numBlocks);
        scan_kernel_eff_large3 << < numBlocks, SECTION_SIZE >> > (result, S, n);
        tmps->free_points(S);
    } else {
        scan_kernel_eff << < numBlocks, SECTION_SIZE >> > (source, result, n);
    }
}


void inclusive_scan_nodes(int *source, int *result, int n, TmpMalloc *tmps) {
    int numBlocks = n / SECTION_SIZE;
    if (n % SECTION_SIZE) numBlocks++;

    if (n > SECTION_SIZE) {
        int *S = tmps->malloc_nodes();
        scan_kernel_eff_large1 << < numBlocks, SECTION_SIZE >> > (source, result, S, n);
        inclusive_scan(S, S, numBlocks);
        scan_kernel_eff_large3 << < numBlocks, SECTION_SIZE >> > (result, S, n);
        tmps->free_nodes(S);

    } else {
        scan_kernel_eff << < numBlocks, SECTION_SIZE >> > (source, result, n);
        gpuErrchk(hipPeekAtLastError());
    }
}

void inclusive_scan_any(int *source, int *result, int n, TmpMalloc *tmps) {
    int numBlocks = n / SECTION_SIZE;
    if (n % SECTION_SIZE) numBlocks++;

    if (n > SECTION_SIZE) {
        int *S = tmps->malloc_any(numBlocks);
        scan_kernel_eff_large1 << < numBlocks, SECTION_SIZE >> > (source, result, S, n);
        inclusive_scan_any(S, S, numBlocks, tmps);
        scan_kernel_eff_large3 << < numBlocks, SECTION_SIZE >> > (result, S, n);
        tmps->free_any(S, numBlocks);

    } else {
        scan_kernel_eff << < numBlocks, SECTION_SIZE >> > (source, result, n);
        gpuErrchk(hipPeekAtLastError());
    }
}

void inclusive_scan_cpu(int *d_x, int *d_y, int n) {
    int *h_x = new int[n];
    int *h_y = new int[n];
    hipMemcpy(h_y, d_y, n * sizeof(int), hipMemcpyDeviceToHost);

    int tmp = 0;
    for (int i = 0; i < n; i++) {
        tmp += h_y[i];
        h_x[i] = tmp;
    }

    hipMemcpy(d_x, h_x, n * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    delete h_x;
    delete h_y;
}


void inclusive_scan_async(int *x, int *y, int n, hipStream_t stream) {
    int numBlocks = n / BLOCK_WIDTH;
    if (n % BLOCK_WIDTH) numBlocks++;

    if (n > SECTION_SIZE) {
        int *S;
        hipMalloc((void **) &S, (n / SECTION_SIZE) * sizeof(int));
        scan_kernel_eff_large1 << < numBlocks, BLOCK_WIDTH, 0, stream >> > (x, y, S, n);
        inclusive_scan_async(S, S, n / SECTION_SIZE, stream);
        scan_kernel_eff_large3 << < numBlocks, BLOCK_WIDTH, 0, stream >> > (y, S, n);
        hipFree(S);
    } else {
        scan_kernel_eff << < numBlocks, BLOCK_WIDTH, 0, stream >> > (x, y, n);
    }
}

void populate(int *parents, int *cells, int *counts, int *dim_start, int *dims, int c, int d, int n) {
    int lvl_size = c - c * 1 / 3;
    int prev_lvl_size = 0;
    int prev_count = 0;

    for (int i = 0; i < d; i++) {
        dims[i] = d - i;
        dim_start[i] = prev_count;
        int p = -1;
        for (int j = 0; j < lvl_size; j++) {
            p += j % 3 == 2 ? 0 : 1;

            if (i == 0) {
                parents[j + prev_count] = -1;
            } else {
                parents[j + prev_count] = prev_count - prev_lvl_size + p;
            }
        }
        prev_count += lvl_size;
        prev_lvl_size = lvl_size;
        lvl_size *= 1.5;
    }

    for (int i = 0; i < d; i++) {
        int r_count = 0;
        int c_no = 0;
        for (int j = 0; j < ((i < d - 1 ? dim_start[i + 1] : n) - dim_start[i]); j++) {
            int m = (i == 0 ? c * 1 / 3 : c - 2);
            if (i != 0 && j % 3 != 2) {
                r_count = 0;
                c_no = 0;
            }
            while (r_count < m && rand() % c < m) {
                r_count++;
                c_no++;
            }

            cells[dim_start[i] + j] = c_no + 1;

            c_no++;
        }
    }

    for (int j = 0; j < dim_start[d - 1]; j++) {
        counts[j] = 0;
    }

    for (int j = dim_start[d - 1]; j < n; j++) {
        int count = rand() % 10 * rand() % 10 + 1;
        counts[j] = count;
        int p = parents[j];
        while (p != -1) {
            counts[p] += count;
            p = parents[p];
        }
    }
}

void print_scy_tree(int *parents, int *cells, int *counts, int *dim_start, int *dims, int d, int n) {

    printf("r:  %d/%d\n", cells[0], counts[0]);
    if (d == 0)
        return;

    int *leaf_count = new int[n];

    for (int i = 0; i < n; i++)
        leaf_count[i] = 0;

    for (int i = dim_start[d - 1]; i < n; i++) {
        leaf_count[i] = 0;
        int p = i;
        while (p > 0) {
            leaf_count[p]++;
            p = parents[p];
        }
    }
    for (int i = 0; i < d; i++) {
        printf("%d: ", dims[i]);
        for (int j = dim_start[i]; j < ((i < (d - 1)) ? dim_start[i + 1] : n); j++) {

            if (cells[j] < 100) printf(" ");
            if (cells[j] < 10) printf(" ");
            printf("%d/%d ", cells[j], counts[j]);
            if (counts[j] < 100 && counts[j] > -10) printf(" ");
            if (counts[j] < 10 && counts[j] > -1) printf(" ");

            for (int k = 0; k < leaf_count[j] - 1; k++) {
                printf("        ", cells[j], counts[j]);
            }
        }
        printf("\n");
    }
}

int get_size(int c, int d) {
    int lvl_size = c - c * 1 / 3;
    int prev_count = 0;

    for (int i = 0; i < d; i++) {
        prev_count += lvl_size;
        lvl_size *= 1.5;
    }
    return prev_count;
}

void print_array_range(int *x, int start, int end) {
    for (int i = start; i < end; i++) {
        printf("%d ", (int) x[i]);
    }
    printf("\n\n");
}

void print_array(int *x, int n) {
    int left = 400;
    int right = 400;

    if (n <= left + right) {
        for (int i = 0; i < n; i++) {
            if (x[i] < 10 && x[i] > -1)
                printf(" ");
            if (x[i] < 100 && x[i] > -10)
                printf(" ");
            printf("%d ", (int) x[i]);
        }
    } else {
        for (int i = 0; i < left; i++) {
            printf("%d ", (int) x[i]);
        }
        printf(" ... ");
        for (int i = n - right; i < n; i++) {
            printf("%d ", (int) x[i]);
        }
    }
    printf("\n\n");
}

void print_array(vector<int> x, int n) {
    int left = 400;
    int right = 400;

    if (n <= left + right) {
        for (int i = 0; i < n; i++) {
            printf("%d ", (int) x[i]);
        }
    } else {
        for (int i = 0; i < left; i++) {
            printf("%d ", (int) x[i]);
        }
        printf(" ... ");
        for (int i = n - right; i < n; i++) {
            printf("%d ", (int) x[i]);
        }
    }
    printf("\n\n");
}

void print_array(float *x, int n) {
    int left = 30;
    int right = 10;

    if (n <= left + right) {
        for (int i = 0; i < n; i++) {
            printf("%f ", (float) x[i]);
        }
    } else {
        for (int i = 0; i < left; i++) {
            printf("%f ", (float) x[i]);
        }
        printf(" ... ");
        for (int i = n - right; i < n; i++) {
            printf("%f ", (float) x[i]);
        }
    }
    printf("\n\n");
}

void print_array(thrust::device_vector<int> x, int n) {
    int left = 30;
    int right = 10;

    if (n <= left + right) {
        for (int i = 0; i < n; i++) {
            printf("%d ", (int) x[i]);
        }
    } else {
        for (int i = 0; i < left; i++) {
            printf("%d ", x[i]);
        }
        printf(" ... ");
        for (int i = n - right; i < n; i++) {
            printf("%d ", x[i]);
        }
    }
    printf("\n\n");
}


int get_incorrect(int *array_1, int *array_2, int n) {
    int count = 0;
    for (int i = 0; i < n; i++) {
        if (array_1[i] != array_2[i]) {
            count++;
        }
    }
    return count;
}


float v_mean(std::vector<float> v) {
    //https://stackoverflow.com/questions/28574346/find-average-of-input-to-vector-c
    return accumulate(v.begin(), v.end(), 0.0) / v.size();
}


vector<float> m_get_col(vector <vector<float>> m, int i) {
    vector<float> col;
    for (int j = 0; j < m.size(); j++) {
        col.push_back(m[j][i]);
    }
    return col;
}

float v_min(std::vector<float> v) {
    float min = std::numeric_limits<float>::infinity();
    for (int i = 0; i < v.size(); i++) {
        if (v[i] < min) {
            min = v[i];
        }
    }
    return min;
}

float v_max(std::vector<float> v) {
    float max = -100000.;
    for (int i = 0; i < v.size(); i++) {
        if (v[i] > max) {
            max = v[i];
        }
    }
    return max;
}

int v_max(std::vector<int> v) {
    int max = -100000;
    for (int i = 0; i < v.size(); i++) {
        if (v[i] > max) {
            max = v[i];
        }
    }
    return max;
}


void m_normalize(std::vector <std::vector<float>> &m) {

    float *min = new float[m[0].size()];
    float *max = new float[m[0].size()];

    for (int j = 0; j < m[0].size(); j++) {
        min[j] = 100000.;
        max[j] = -100000.;
    }

    for (int i = 0; i < m.size(); i++) {
        for (int j = 0; j < m[0].size(); j++) {
            min[j] = min[j] < m[i][j] ? min[j] : m[i][j];
            max[j] = max[j] > m[i][j] ? max[j] : m[i][j];
        }
        printf("finding min/max: %d%%\r", int(((i + 1) * 100) / m.size()));
    }
    printf("finding min/max: 100%%\n");

    for (int i = 0; i < m.size(); i++) {
        for (int j = 0; j < m[0].size(); j++) {
            m[i][j] = max[j] != min[j] ? (m[i][j] - min[j]) / (max[j] - min[j]) : 0;
        }
        printf("normalizing: %d%%\r", int(((i + 1) * 100) / m.size()));
    }
    printf("normalizing: 100%%\n");
}

template<class T>
vector <T> clone(vector <T> v_old) {
    vector <T> v_clone(v_old);
    return v_clone;
}

void zero(int *array, int n) {
    for (int i = 0; i < n; i++)
        array[i] = 0;
}


bool subspace_of(vector<int> subspace, vector<int> subspace_mark) {
    int i = 0;
    int j = 0;
    while (j < subspace_mark.size() && i < subspace.size()) {
        if (subspace[i] == subspace_mark[j]) {
            i++;
            j++;
        } else {
            j++;
        }
    }
    return i == subspace.size();
}

bool vec_cmp::operator()(const vector<int> &a, const vector<int> &b) const {
    int i = a.size() - 1;
    int j = b.size() - 1;
    while (a[i] == b[j]) {
        i--;
        j--;
        if (i < 0 || j < 0) {
            return i < j;
        }
    }

    return a[i] < b[j];
}

void join(map <vector<int>, vector<int>, vec_cmp> &result, vector<int> &clustering, vector<int> subspace, int min_size,
          float r) {

    int clustering_max = v_max(clustering);
    if (clustering_max < 0) {
        return;
    }

    int n = clustering.size();

    map<int, int> sizes;

    for (int i = 0; i < n; i++) {
        int cluster = clustering[i];
        if (cluster >= 0) {
            if (sizes.count(cluster)) {
                sizes[cluster]++;
            } else {
                sizes.insert(pair<int, int>(cluster, 1));
            }
        }
    }

    for (int i = 0; i < n; i++) {
        int cluster = clustering[i];
        if (cluster >= 0 && sizes[cluster] < min_size) {
            clustering[i] = -1;
        }
    }

    for (pair <vector<int>, vector<int>> subspace_clustering : result) {

        vector<int> subspace_H = subspace_clustering.first;
        vector<int> clustering_H = subspace_clustering.second;

        if (subspace_of(subspace, subspace_H)) {

            map<int, int> sizes_H;
            set<int> to_be_removed;
            for (int cluster_id: clustering_H) {
                if (cluster_id >= 0) {
                    if (sizes_H.count(cluster_id)) {
                        sizes_H[cluster_id]++;
                    } else {
                        sizes_H.insert(pair<int, int>(cluster_id, 1));
                    }
                }
            }

            for (int i = 0; i < n; i++) {
                int cluster = clustering[i];
                int cluster_H = clustering_H[i];
                if (cluster >= 0 && cluster_H >= 0 && sizes[cluster] * r < sizes_H[cluster_H]) {
                    to_be_removed.insert(cluster);
                }
            }

            for (int i = 0; i < n; i++) {
                int cluster = clustering[i];
                if (cluster >= 0 &&
                    to_be_removed.find(cluster) != to_be_removed.end()) {
                    clustering[i] = -1;
                }
            }
        }
    }

    clustering_max = v_max(clustering);
    if (clustering_max >= 0) {
        if (result.count(subspace)) {
            vector<int> clustering_old = result[subspace];
            int m = v_max(clustering_old);
            for (int i = 0; i < n; i++) {
                if (clustering[i] == -2) {
                    clustering_old[i] = clustering[i];
                } else if (clustering[i] >= 0) {
                    clustering_old[i] = m + 1 + clustering[i];
                }
            }
            result[subspace] = clustering_old;
        } else {
            result.insert(pair < vector < int > , vector < int >> (subspace, clustering));
        }
    }
}


__global__
void join_count_kernel(int *d_sizes, int *d_clustering, int n) {
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        int cluster = d_clustering[i];
        if (cluster >= 0) {
            atomicAdd(&d_sizes[cluster], 1);
        }
    }
}

__global__
void join_erease_kernel(int *d_sizes, int *d_clustering, int n, int min_size) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        int cluster = d_clustering[i];
        if (cluster >= 0 && d_sizes[cluster] < min_size) {
            d_clustering[i] = -1;
        }
    }
}

__global__
void
join_marke_remove_kernel(int *d_to_be_removed, int *d_sizes, int *d_clustering, int *d_sizes_H, int *d_clustering_H,
                         int n, float r) {

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        int cluster = d_clustering[i];
        int cluster_H = d_clustering_H[i];
        if (cluster >= 0 && cluster_H >= 0 && d_sizes[cluster] * r < d_sizes_H[cluster_H]) {
            d_to_be_removed[cluster] = 1;
        }
    }
}

__global__
void join_remove_kernel(int *d_to_be_removed, int *d_sizes, int *d_clustering, int n) {

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        int cluster = d_clustering[i];
        if (cluster >= 0 && d_to_be_removed[cluster]) {
            d_clustering[i] = -1;
        }
    }
}

__global__
void copy_if_positive(int *d_clustering_old, int *d_clustering, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (d_clustering[i] >= 0) {
            d_clustering_old[i] = d_clustering[i];
        }
    }
}

__global__
void v_max_gpu_kernel(int *d_max, int *d_array, int n) {
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        atomicMax(&d_max[0], d_array[i]);
    }
}

int v_max_gpu(int *d_array, int n) {
    int BLOCK_SIZE = 512;
    int number_of_threads = min(n, BLOCK_SIZE);

    int *d_max;
    hipMalloc(&d_max, sizeof(int));
    hipMemset(d_max, -1, sizeof(int));

    v_max_gpu_kernel << < 1, number_of_threads >> > (d_max, d_array, n);

    int h_max;
    hipMemcpy(&h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_max);
    return h_max;
}

int v_max_gpu(int *d_array, int n, TmpMalloc *tmps) {
    int BLOCK_SIZE = 512;
    int number_of_threads = min(n, BLOCK_SIZE);

    int *d_max = tmps->malloc_one();
    hipMemset(d_max, -1, sizeof(int));

    v_max_gpu_kernel << < 1, number_of_threads >> > (d_max, d_array, n);

    int h_max;
    hipMemcpy(&h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
    tmps->free_one(d_max);
    return h_max;
}

void
join_gpu(map<vector<int>, int *, vec_cmp> &result, int *d_clustering,
         vector<int> subspace, int min_size, float r, int n, TmpMalloc *tmps) {
    tmps->reset_counters();

    int BLOCK_SIZE = 512;
    int number_of_blocks = n / BLOCK_SIZE;
    if (n % BLOCK_SIZE) number_of_blocks++;
    int number_of_threads = min(n, BLOCK_SIZE);

    int *d_sizes = tmps->get_int_array(tmps->int_array_counter++, n);
    hipMemset(d_sizes, 0, n * sizeof(int));

    join_count_kernel << < 1, number_of_threads >> > (d_sizes, d_clustering, n);

    join_erease_kernel << < number_of_blocks, number_of_threads >> > (d_sizes, d_clustering, n, min_size);


    int *d_clustering_H;
    int *d_sizes_H = tmps->get_int_array(tmps->int_array_counter++, n);
    int *d_to_be_removed = tmps->get_int_array(tmps->int_array_counter++, n);

    for (pair<vector<int>, int *> subspace_clustering : result) {

        vector<int> subspace_H = subspace_clustering.first;
        d_clustering_H = subspace_clustering.second;


        nvtxRangePushA("subspace_of");
        bool sub_of = subspace_of(subspace, subspace_H);
        nvtxRangePop();
        if (sub_of) {

            hipMemset(d_sizes_H, 0, n * sizeof(int));
            hipMemset(d_to_be_removed, 0, n * sizeof(int));

            join_count_kernel << < 1, number_of_threads >> > (d_sizes_H, d_clustering_H, n);

            join_marke_remove_kernel << < number_of_blocks, number_of_threads >> >
            (d_to_be_removed, d_sizes, d_clustering, d_sizes_H, d_clustering_H, n, r);

            join_remove_kernel << < number_of_blocks, number_of_threads >> >
            (d_to_be_removed, d_sizes, d_clustering, n);
        }
    }


    int clustering_max = v_max_gpu(d_clustering, n);
    if (clustering_max >= 0) {
        if (result.count(subspace)) {
            int *d_clustering_old = result[subspace];

            copy_if_positive << < number_of_blocks, number_of_threads >> > (d_clustering_old, d_clustering, n);
            tmps->free_points(d_clustering);
            result[subspace] = d_clustering_old;
        } else {
            result.insert(pair < vector < int > , int * > (subspace, d_clustering));
        }
    } else {
        tmps->free_points(d_clustering);
    }
}

void
join_gpu1(map <vector<int>, vector<int>, vec_cmp> &result, vector<int> &clustering, int *d_clustering,
          vector<int> subspace, int min_size,
          float r, int n) {

    int BLOCK_SIZE = 512;
    int number_of_blocks = n / BLOCK_SIZE;
    if (n % BLOCK_SIZE) number_of_blocks++;
    int number_of_threads = min(n, BLOCK_SIZE);

    int *d_sizes;
    hipMalloc(&d_sizes, n * sizeof(int));
    hipMemset(d_sizes, 0, n * sizeof(int));

    join_count_kernel << < 1, number_of_threads >> > (d_sizes, d_clustering, n);

    join_erease_kernel << < number_of_blocks, number_of_threads >> > (d_sizes, d_clustering, n, min_size);

    int *d_subspace_H;
    int *d_clustering_H;
    hipMalloc(&d_clustering_H, n * sizeof(int));

    int *d_sizes_H;
    hipMalloc(&d_sizes_H, n * sizeof(int));

    int *d_to_be_removed;
    hipMalloc(&d_to_be_removed, n * sizeof(int));

    for (pair <vector<int>, vector<int>> subspace_clustering : result) {

        vector<int> subspace_H = subspace_clustering.first;
        vector<int> clustering_H = subspace_clustering.second;
        hipMemcpy(d_clustering_H, clustering_H.data(), n * sizeof(int), hipMemcpyHostToDevice);

        if (subspace_of(subspace, subspace_H)) {

            hipMemset(d_sizes_H, 0, n * sizeof(int));
            hipMemset(d_to_be_removed, 0, n * sizeof(int));

            join_count_kernel << < 1, number_of_threads >> > (d_sizes_H, d_clustering_H, n);

            join_marke_remove_kernel << < number_of_blocks, number_of_threads >> >
            (d_to_be_removed, d_sizes, d_clustering, d_sizes_H, d_clustering_H, n, r);

            join_remove_kernel << < number_of_blocks, number_of_threads >> >
            (d_to_be_removed, d_sizes, d_clustering, n);
        }
    }
    hipMemcpy(clustering.data(), d_clustering, n * sizeof(int), hipMemcpyDeviceToHost);

    int clustering_max = v_max(clustering);
    int clustering_max_other = v_max_gpu(d_clustering, n);
    if (clustering_max != clustering_max_other)
        printf("%d!=%d\n", clustering_max, clustering_max_other);
    if (clustering_max >= 0) {
        if (result.count(subspace)) {
            vector<int> clustering_old = result[subspace];
            int *d_clustering_old;
            hipMalloc(&d_clustering_old, n * sizeof(int));
            hipMemcpy(d_clustering_old, clustering_old.data(), n * sizeof(int), hipMemcpyHostToDevice);

            copy_if_positive << < number_of_blocks, number_of_threads >> > (d_clustering_old, d_clustering, n);
            hipMemcpy(clustering_old.data(), d_clustering_old, n * sizeof(int), hipMemcpyDeviceToHost);

            result[subspace] = clustering_old;
        } else {
            result.insert(pair < vector < int > , vector < int >> (subspace, clustering));
        }
    }
}